#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <iostream>

template <typename T, int A_ROWS, int A_COLS>
MyLA::Matrix<T, ROWS, COLS>(T defaultVal) {
    for(int i = 0; i < A_ROWS*A_COLS; i++) {
        hostData[i] = defaultVal;
    }
}

// ROWS = y
// COLS = x
template <typename T, int A_ROWS, int A_COLS>
template <int B_ROWS, int B_COLS>
MyLA::Matrix<T, A_ROWS, A_COLS> MyLA::Matrix<T, A_ROWS, A_COLS>::operator+(const Matrix<T, B_ROWS, B_COLS>& hostB) const {
    assert(A_ROWS == B_ROWS && A_COLS == B_COLS);
    MyLA::Matrix<T, A_ROWS, A_COLS> hostC;
    T* devA;
    T* devB;
    T* devC;
    size_t size = A_ROWS*A_COLS*sizeof(T);

    hipError_t error = hipSuccess;

    error = hipMalloc((void**)&devA, size);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devA for Matrix Addition" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }
    error = hipMalloc((void**)&devB, size);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devB for Matrix Addition" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }
    error = hipMalloc((void**)&devC, size);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devC for Matrix Addition" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }

    error = hipMemcpy(devA, this->getData(), size, hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix A from host to device" << std::endl;
        throw std::runtime_error("Failed to copy host to device memory.");
    }

    error = hipMemcpy(devB, hostB.getData(), size, hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix B from host to device" << std::endl;
        throw std::runtime_error("Failed to copy host to device memory.");
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((A_COLS+threadsPerBlock.x-1)/threadsPerBlock.x, (A_ROWS+threadsPerBlock.y-1)/threadsPerBlock.y);

    matAddKernel<<<numBlocks, threadsPerBlock>>>(devA, devB, devC, A_ROWS, A_COLS);

    error = hipMemcpy(hostC.getData(), devC, size, hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix C from device to host" << std::endl;
        throw std::runtime_error("Failed to copy device to host memory.");
    }

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    return hostC;
}

template <typename T>
__global__ void matAddKernel(T* devA, T* devB, T* devC, int rows, int cols) {
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(row < rows && col < cols) {
        int index = (row*cols) + col;
        devC[index] = devA[index] + devB[index];
    }
}

template<typename T, int A_ROWS, int A_COLS>
template<int B_ROWS, int B_COLS>
MyLA::Matrix<T, A_ROWS, B_COLS> MyLA::Matrix<T, A_ROWS, A_COLS>::operator*(const Matrix<T, B_ROWS, B_COLS>& hostB) const {
    assert(A_COLS == B_COLS);
    MyLA::Matrix<T, A_ROWS, B_COLS> hostC;
    T* devA;
    T* devB;
    T* devC;
    size_t sizeA = A_ROWS*A_COLS*sizeof(T);
    size_t sizeB = B_ROWS*B_COLS*sizeof(T);
    size_t sizeC = A_ROWS*B_COLS*sizeof(T);
    hipError_t error;

    error = hipMalloc((void**)&devA, sizeA);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devA for Matrix Matrix Multiplication" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }

    error = hipMalloc((void**)&devB, sizeB);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devB for Matrix Matrix Multiplication" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }

    error = hipMalloc((void**)&devC, sizeC);
    if(error != hipSuccess) {
        std::cerr << "Failed to allocate devC for Matrix Matrix Multiplication" << std::endl;
        throw std::runtime_error("Failed to allocate device memory.");
    }

    error = hipMemcpy(devA, this->getData(), sizeA, hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix A from host to device" << std::endl;
        throw std::runtime_error("Failed to copy host to device memory.");
    }

    error = hipMemcpy(devB, hostB.getData(), sizeB, hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix B from host to device" << std::endl;
        throw std::runtime_error("Failed to copy host to device memory.");
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((B_COLS+threadsPerBlock.x-1)/threadsPerBlock.x, (A_ROWS+threadsPerBlock.y-1)/threadsPerBlock.y);

    matmatMulKernel<<<numBlocks, threadsPerBlock>>>(devA, devB, devC, A_ROWS, A_COLS, B_COLS);

    error = hipMemcpy(hostC.getData(), devC, sizeC, hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        std::cerr << "Failed to copy Matrix C from device to host" << std::endl;
        throw std::runtime_error("Failed to copy device to host memory.");
    }

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    return hostC;
}

template<typename T>
__global__ void matmatMulKernel(T* devA, T* devB, T* devC, int rowsA, int colsrowsAB, int colsB) {
    int row = (blockIdx.y*blockDim.y) + threadIdx.y;
    int col = (blockIdx.x*blockDim.x) + threadIdx.x;
    if(row < rowsA && col < colsB) {
        T accum = 0;
        for(int i = 0; i < colsrowsAB; i++) {
            accum += devA[(row*colsrowsAB)+i] * devB[(i*colsB)+col];
        }
        int indexC = (row*colsB) + col;
        devC[indexC] = accum;
    }
}
